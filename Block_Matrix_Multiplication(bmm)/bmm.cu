#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block

// Best TILEX : 32 and Best TILEY : 16 
#define TILEX 32
#define TILEY 16

// Best L = min(TILEX,TILEY)
#define L (TILEX <= TILEY ? TILEX : TILEY)

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}

dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}

__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	// Varibale definition needed for loops :
	int p;
	int k;	

	// Main index definition :
	int i = by * TILEY + ty;
	int j = bx * TILEX + tx;
	
	// C'sum Variable :
	float sum = 0;
	
	// Shared memory defnintion :
	__shared__ float as[TILEY][4*L];
	__shared__ float bs[4*L][TILEX];	

	for ( p = 0 ; p < (n/L) ; p = p + 4){
		
		// copy from ad to as (Shared memory) 
		// unrolled loop for best performance :

		if(tx < TILEY){
			as[ty][tx] = ad[((i)<<(m)) + (L*p+tx)];
		}
		
		if(tx < TILEY){
			as[ty][tx + L] = ad[((i)<<(m)) + (L*(p+1)+tx)];
		}
		
		if(tx < TILEY){
			as[ty][tx + 2*L] = ad[((i)<<(m)) + (L*(p+2)+tx)];
		}
		
		if(tx < TILEY){
			as[ty][tx + 3*L] = ad[((i)<<(m)) + (L*(p+3)+tx)];
		}
		
		// copy from bd to bs (Shared memory) 
		// unrolled loop for best performance :

		if (ty < TILEX){
			bs[ty][tx] = bd[((L*p + ty)<<(m)) + (j)];
		}

		if (ty < TILEX){
			bs[ty + L][tx] = bd[((L * (p+1) + ty) << (m)) + (j)];
		}
		
		if (ty < TILEX){
			bs[ty + 2*L][tx] = bd[((L * (p+2) + ty) << (m)) + (j)];
		}
		
		if (ty < TILEX){
			bs[ty + 3*L][tx] = bd[((L * (p+3) + ty) << (m)) + (j)];
		}
		
		// Sync all threads :
		__syncthreads();
		
		// multiplications :
		for (k = 0 ; k < 4 * L ; k++){ 
			sum += as[ ty ][ k ] * bs[ k ][ tx ];
		}
		__syncthreads();
	}
	
	cd [( (i) << (m) ) + ( j )] = sum;
}
